#include "hip/hip_runtime.h"
#include "defs.cuh"
#include "v3.cuh"
#include "particle_system.cuh"

extern __device__ void delta_momentum(v3 * const, particle_system * const, float const, uint const);
extern __device__ void apply_momentum(particle_system * const);

__global__ void cuda_tick(uint const N, float * const m, v3 * const r, v3 * const p, v3 * const dp, float const dt)
{
	v3_set(p + threadIdx.x, threadIdx.x, threadIdx.x, threadIdx.x);

	//__syncthreads();
}

__host__ hipError_t cuda_init(particle_system * const d_ps, v3 * * const d_dp, particle_system const * const ps)
{
	hipError_t status;

	//alloc

	if ((status = hipMalloc((void**)&(d_ps->m), sizeof(float) * ps->N)))
		return status;

	if ((status = hipMalloc((void**)&(d_ps->r), sizeof(v3) * ps->N)))
		return status;

	if ((status = hipMalloc((void**)&d_ps->p, sizeof(v3) * ps->N)))
		return status;

	if ((status = hipMalloc((void**)d_dp, sizeof(v3) * ps->N)))
		return status;

	//copy

	if ((status = hipMemcpy(d_ps->m, ps->m, sizeof(float) * ps->N, hipMemcpyHostToDevice)))
		return status;

	if ((status = hipMemcpy(d_ps->r, ps->r, sizeof(v3) * ps->N, hipMemcpyHostToDevice)))
		return status;

	if ((status = hipMemcpy(d_ps->p, ps->p, sizeof(v3) * ps->N, hipMemcpyHostToDevice)))
		return status;

	return hipSuccess;
}

__host__ hipError_t cuda_launch(particle_system * const ps, v3 * const dp, float const dt)
{
	hipError_t status;

	cuda_tick <<<1, 3>>>(ps->N, ps->m, ps->r, ps->p, dp, dt);

	if ((status = hipGetLastError()))
		return status;

	if ((status = hipDeviceSynchronize()))
		return status;

	return hipSuccess;
}

__host__ hipError_t cuda_sync(particle_system * const ps, particle_system const * const d_ps)
{
	hipError_t status;

	if ((status = hipMemcpy(ps->p, d_ps->p, sizeof(v3) * ps->N, hipMemcpyDeviceToHost)))
		return status;

	return hipSuccess;
}

__host__ hipError_t cuda_deinit(particle_system * const d_ps, v3 * const d_dp)
{
	hipError_t status;

	status = hipDeviceReset();

end:
	hipFree(d_ps->m);
	hipFree(d_ps->r);
	hipFree(d_ps->p);
	hipFree(d_dp);

	return status;
}