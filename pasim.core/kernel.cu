#include "hip/hip_runtime.h"
#include "defs.cuh"
#include "v3.cuh"
#include "particle_system.cuh"

extern __device__ void delta_momentum(v3 * const, particle_system * const, float const, uint const);
extern __device__ void apply_momentum(particle_system * const);

__global__ void cuda_tick(uint const N, float * const m, v3 * const r, v3 * const p, v3 * const dp, float const dt)
{
	v3_set(p + threadIdx.x, threadIdx.x, threadIdx.x, threadIdx.x);

	//__syncthreads();
}

__host__ hipError_t cuda_init(particle_system * const ps)
{
	hipError_t status;

	if ((status = hipMalloc((void**)&ps->dev_m, sizeof(float) * ps->N)))
		return status;

	if ((status = hipMalloc((void**)&ps->dev_r, sizeof(v3) * ps->N)))
		return status;

	if ((status = hipMalloc((void**)&ps->dev_p, sizeof(v3) * ps->N)))
		return status;

	if ((status = hipMalloc((void**)&ps->dev_dp, sizeof(v3) * ps->N)))
		return status;

	return hipSuccess;
}

__host__ hipError_t cuda_sync_dev(particle_system const * const ps, float const * const m, v3 const * const r, v3 const * const p)
{
	hipError_t status;

	if ((status = hipMemcpy(ps->dev_m, m, sizeof(float) * ps->N, hipMemcpyHostToDevice)))
		return status;

	if ((status = hipMemcpy(ps->dev_r, r, sizeof(v3) * ps->N, hipMemcpyHostToDevice)))
		return status;

	if ((status = hipMemcpy(ps->dev_p, p, sizeof(v3) * ps->N, hipMemcpyHostToDevice)))
		return status;

	return hipSuccess;
}

__host__ hipError_t cuda_launch(particle_system * const ps, float const dt)
{
	hipError_t status;

	cuda_tick<<<1, 3>>>(ps->N, ps->dev_m, ps->dev_r, ps->dev_p, ps->dev_dp, dt);

	if ((status = hipGetLastError()))
		return status;

	if ((status = hipDeviceSynchronize()))
		return status;

	return hipSuccess;
}

__host__ hipError_t cuda_sync_host(particle_system * const ps)
{
	return hipMemcpy(ps->r, ps->dev_r, sizeof(v3) * ps->N, hipMemcpyDeviceToHost);
}

__host__ hipError_t cuda_deinit(particle_system * const ps)
{
	hipError_t status;

	status = hipDeviceReset();

end:
	hipFree(ps->dev_m);
	hipFree(ps->dev_r);
	hipFree(ps->dev_p);
	hipFree(ps->dev_dp);

	return status;
}