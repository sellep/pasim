#include "hip/hip_runtime.h"
#include "kernel_base.cuh"

__global__ void apply_momentum_naive_2(
    float4       * const bs,
    float3       * const ps,
    float          const N,
    float          const dt)
{
    float4 bi;
    float3 pi;
    uint i;

    for (i = blockDim.x * blockIdx.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x)
    {
		bi = bs[i];
		pi = ps[i];

		bi.x = pi.x * dt / bi.w;
		bi.y = pi.y * dt / bi.w;
		bi.z = pi.z * dt / bi.w;

		bs[i] = bi;
    }
}